#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <chrono>
#include <thread>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

using namespace std;

__constant__ int RACE_DISTANCE = 100;
__constant__ int MAX_SPEED = 5;

class Runner {
    private:
        int position;
        int speed;
        int index;

    public:
        void init(int index){
            this->position = 0;
            this->speed = 0;
            this->index = index;
        }

        void setSpeed(int speed){
            this->speed = speed;
        }

        __device__ void increasePosition(){
            if(this->position >= RACE_DISTANCE){
                this->position += MAX_SPEED;
                return;
            }
            this->position += this->speed;
        }

        __device__ __host__ int getPosition() const{
            return this->position;
        }

        int getIndex(){
            return this->index;
        }
};


__host__ __device__ bool operator<(const Runner &o1, const Runner &o2) 
{
   return o1.getPosition() > o2.getPosition();
}

__global__ void calculate_race_positions(
    Runner * d_runners,
    bool *is_any_finish,
    bool *is_any_racing
) {
    d_runners[threadIdx.x].increasePosition();

    if( d_runners[threadIdx.x].getPosition() >= RACE_DISTANCE ){
        *is_any_finish = true;
    }else{
        *is_any_racing = true;
    }
}

class RaceService{
    private:
        bool *d_is_any_finish, *d_is_any_racing;
        bool is_any_finish;
        bool is_any_racing;
        bool is_finish_printed;

        int participant_count;
        int required_mem_size;

        Runner *runners;
        Runner *d_runners;

        void fillSpeedsRandom(int size){
            for(int i = 0; i < size; i++){
                this->runners[i].setSpeed(1 + (rand() % 5));
            }
        }

        void randomizeVelocities(){
            fillSpeedsRandom(this->participant_count);
        }

    public:
        void init(int number_of_contestants){
            this->participant_count = number_of_contestants;
            this->required_mem_size = sizeof(Runner) * number_of_contestants;
            this->runners = (Runner *)malloc(this->required_mem_size);

            for(int i = 0; i < this->participant_count; i++){
                this->runners[i].init(i+1);
            }

            this->presetFlags();
            this->is_finish_printed = false;

            hipMalloc((void **)&d_runners, this->required_mem_size);

            hipMalloc((void **)&d_is_any_finish, sizeof(bool));
            hipMalloc((void **)&d_is_any_racing, sizeof(bool));
        }

        void presetFlags(){
            this->is_any_finish = false;
            this->is_any_racing = false;
        }

        void cleanup(){
            free(this->runners);
            hipFree(d_runners); hipFree(d_is_any_finish); hipFree(d_is_any_racing);
        }

        void printPositions(){
            cout << "Bitiş çizgisine ilk koşucu ulaştı." << endl;
            for(int i = 0; i < this->participant_count; i++){
                int position = this->runners[i].getPosition();
                if(position >= 100){
                    position = 100;
                }
                cout << i+1 << "- " << position << ". metrede" << endl;
            }
            cout << "########### YARIŞ BİTTİ ###########" << endl;
        }

        void printIndexes(thrust::host_vector<Runner> h_runners){
            for(int i=0; i < h_runners.size(); i++)
                cout << i+1 << ". :" << h_runners[i].getIndex() << " Numarali Kosucu " << endl;
            cout << endl;
        }

        void move_to_next_position(){
            this->randomizeVelocities();
            this-> presetFlags();

            hipMemcpy(
                d_runners,
                runners,
                required_mem_size,
                hipMemcpyHostToDevice
            );
            hipMemcpy(
                d_is_any_finish,
                &is_any_finish,
                sizeof(bool),
                hipMemcpyHostToDevice
            );
            hipMemcpy(
                d_is_any_racing,
                &is_any_racing,
                sizeof(bool),
                hipMemcpyHostToDevice
            );

            calculate_race_positions<<<1, this->participant_count>>>(
                d_runners,
                d_is_any_finish,
                d_is_any_racing
            );

            hipMemcpy(
                &is_any_finish,
                d_is_any_finish,
                sizeof(bool),
                hipMemcpyDeviceToHost
            );
            hipMemcpy(
                &is_any_racing,
                d_is_any_racing,
                sizeof(bool),
                hipMemcpyDeviceToHost
            );
            hipMemcpy(
                runners,
                d_runners,
                required_mem_size,
                hipMemcpyDeviceToHost
            );
        }

        void sort_positions(){
            thrust::device_vector<Runner> cd;
            thrust::host_vector<Runner> ch;
            for(int i = 0; i < this->participant_count; i++){
                ch.push_back(this->runners[i]);
            }
            cd = ch;
            thrust::sort(cd.begin(), cd.end());
            ch = cd;
            printIndexes(ch);
        }

        void start(){
            while (true){
                move_to_next_position();
                if(this->is_any_finish && !this->is_finish_printed){
                    printPositions();
                    this->is_finish_printed = true;
                }
                if(!this->is_any_racing){
                    sort_positions();
                    break;
                }
                this_thread::sleep_for(chrono::milliseconds(1000));
            }
        }
};

int main(void){
    RaceService race;
    race.init(100);
    race.start();
    race.cleanup();
}
